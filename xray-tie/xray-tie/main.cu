#include "hip/hip_runtime.h"
//	main.cpp
//  orginal author: Diivanand Ramalingam
//  original institution: Computational Optical Imaging Lab at UC Berkeley (Prof. Laura Waller's Lab)

#include <ctime>
#include <cstdio>
#include <cstdlib>
#include <string>
#include <iostream>
#include "tiff_io-win.h"
#include "toolbox.h"
#include "pointwise_matrix_ops.h"
#include "fourier_tools.h"

//These constants may already be in <cmath> so I should switch the using those in the code but for now I'm using these
#define E 2.71828182845904523536028747135266249775724709369995 //only used e for testing log base e at some point so could remove this now
#define PI  3.1415926535897932384626433832795028841971693993751058209749445923078164062862089986280348253421170679
//Not all GPUS can handle large block sizes which is why it's only 16 for now
#define BLOCKSIZEX 16 //should increase this to 32,64,128, etc. to see potentially better performance gains!
#define BLOCKSIZEY 16 //should increase this to 32,64,128, etc. to see potentially better performance gains!

//device function delclaration
__global__ void xrayTIEHelperKernel(hipfftComplex *denominator_dev, float *freq_vector_dev, int N, float R2, float delta, float Mag, float mu, float reg);

//host function declarations
void stateArguments(float IinVal, float Mag, float R2, float mu, float delta, float ps, float reg);
hipError_t calculateThickness(float *output, float *image, int height, int width, float IinVal, float Mag, float R2, float mu, float delta, float ps, float reg);

/* Height is the number of rows (x) and Width is the number of columns (y)*/
int main(int argc, char **argv)
{
	if(argc != 13){
		printf("Incorrect number of arguments. Usage: ./tie input_folder output_folder prefix start_num end_num Iin Mag R2 mu delta ps reg\n");
		quitProgramPrompt(0);
		return 1;
	}else {
		char *srcFolder = argv[1];
		char *destFolder = argv[2];
		char *prefix = argv[3];
		int start = atoi(argv[4]);
		int end = atoi(argv[5]);
		int numFiles = end - start + 1;
		char **filenames = getFilenames(srcFolder, prefix, start, end);
		char **outfilenames = getFilenames(destFolder, prefix, start, end);
		TIFFSetWarningHandler(NULL);
		//IinVal, Mag, R2, mu, delta, ps
		float IinVal = atof(argv[6]);
		float Mag = atof(argv[7]);
		Mag = 1.0; //Right now algorithm doesn't work for Mag other than 1.0, so for now Mag argument isn't supported.
		float R2 = atof(argv[8]);
		float mu = atof(argv[9]);
		float delta = atof(argv[10]);
		float ps = atof(argv[11]);
		float reg = atof(argv[12]);

		stateArguments(IinVal, Mag, R2, mu, delta, ps, reg);

		TiffIO* tiff_io = new TiffIO();
		int width;
		int height;
		printf("Processing Input Files: \n");
		for(int i = 0;i < numFiles; i++) {
			float **image;
			//read iamge
			image = tiff_io->read16bitImage(filenames[i], &width, &height);

			if(!image){
				printf("Error reading image\n");
			}else {
				//convert image to 1D for CUDA processing
				float *image1D = toFloatArray(image, width, height);
				float *output = (float *) malloc(sizeof(float) * width * height);
				
				printf("\nProcessing file %s\n", filenames[i]);
				//Process Image
				
				calculateThickness(output, image1D, height, width, IinVal, Mag, R2, mu, delta, ps, reg);
				
				
				//End Processing of Image
				//convert image back to 2D for outputting
				float *image1DOut = image1D;

				image = toFloat2D(image1DOut, width, height);
				//output image
				printf("\nFile Processed. Outputting to %s\n", outfilenames[i]);
				tiff_io->write16bitImage(image, outfilenames[i], width, height);
				

				//free memory
				free(image1D);
				free(output);
				delete image;
			}
		}
		delete tiff_io;
		quitProgramPrompt(true);
		return 0;
	}
}

void stateArguments(float IinVal, float Mag, float R2, float mu, float delta, float ps, float reg)
{
	std::cout << "Input Argument Values:" << std::endl;
	std::cout << "IinVal: " << IinVal << std::endl;
	std::cout << "Mag: " << Mag << std::endl;
	std::cout << "R2: " << R2 << " mm" << std::endl;
	std::cout << "mu: " << mu << " mm^-1" << std::endl;
	std::cout << "delta: " << delta << std::endl;
	std::cout << "ps: " << ps << " mm" << std::endl;
	std::cout << "reg: " << reg << std::endl;
}

/*
Calculates thickness according to Paganin Phase paper algorithm: http://www.ncbi.nlm.nih.gov/pubmed/12000561
*/
hipError_t calculateThickness(float* output, float *image, int height, int width, float IinVal, float Mag, float R2, float mu, float delta, float ps, float reg)
{
	hipError_t cudaStatus;
	hipfftResult cufftStatus;
	hipfftHandle plan = 0;
	if(height != width){
		fprintf(stderr, "Only works on square matrices whose dimension is a power of two!\n", cudaStatus);
        goto thickness_end;
	}
	//declare and initialize variables used when calling CUDA kernels
	int size = height * width;
	int block_size_x = BLOCKSIZEX;
    int block_size_y = BLOCKSIZEY;
    dim3 dimBlock(block_size_x, block_size_y);
    dim3 dimGrid (height/dimBlock.x, width/dimBlock.y);
	int N = width;
	//Handle N not multiple of block_size_x or block_size_y but this shouldn't be the case since N power of 2
	//And blocksize should always be a power of 2 for both correctness and efficiency
    if (height % block_size_x !=0 ) dimGrid.x+=1;
    if (width % block_size_y !=0 ) dimGrid.y+=1;

	std::cout << "Calculating Thickness..." << std::endl;
	std::clock_t begin = std::clock();
	//Code begins here

	//declare device pointers
	float *int_seq_dev = 0;
	float *freq_vector_dev = 0;
	float *image_dev = 0;
	float *output_dev = 0;
	hipfftComplex *image_complex_dev = 0;
	hipfftComplex *fft_output_dev = 0;
	hipfftComplex *fft_shifted_output_dev = 0;
	hipfftComplex *ifft_shifted_input_dev = 0;
	hipfftComplex *ifft_output_dev = 0;
	hipfftComplex *denominator_dev = 0;

	//Start memory allocation of device vectors and convert/copy input image to complex device vector

	//Allocate memory for 10 device vectors (potential speedup to be gained by reducing the number of device vectors used)
	cudaStatus = hipMalloc((void**)&int_seq_dev, N * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for int_seq_dev failed!");
        goto thickness_end;
    }
	cudaStatus = hipMalloc((void**)&freq_vector_dev, N * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for freq_vector_dev failed!");
        goto thickness_end;
    }
	cudaStatus = hipMalloc((void**)&image_dev, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for image_dev failed!");
        goto thickness_end;
    }
	cudaStatus = hipMalloc((void**)&output_dev, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for output_dev failed!");
        goto thickness_end;
    }
	cudaStatus = hipMalloc((void**)&image_complex_dev, size * sizeof(hipfftComplex));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for image_complex_dev failed!");
        goto thickness_end;
    }
	cudaStatus = hipMalloc((void**)&fft_output_dev, size * sizeof(hipfftComplex));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for fft_output_dev failed!");
        goto thickness_end;
    }
	cudaStatus = hipMalloc((void**)&fft_shifted_output_dev, size * sizeof(hipfftComplex));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for fft_shifted_output_dev failed!");
        goto thickness_end;
    }
	cudaStatus = hipMalloc((void**)&ifft_shifted_input_dev, size * sizeof(hipfftComplex));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for ifft_shifted_input_dev failed!");
        goto thickness_end;
    }
	cudaStatus = hipMalloc((void**)&ifft_output_dev, size * sizeof(hipfftComplex));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for ifft_output_dev failed!");
        goto thickness_end;
    }
	cudaStatus = hipMalloc((void**)&denominator_dev, size * sizeof(hipfftComplex));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for denominator_dev failed!");
        goto thickness_end;
    }

	//copy input image host vector to device vector and scale by magnification
	cudaStatus = hipMemcpy(image, image_dev, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy for image_dev failed! Error Code: %d", cudaStatus);
        goto thickness_end;
    }
	pointwiseRealScaleRealMatrix<<<dimGrid, dimBlock>>>(image_dev, image_dev, Mag*Mag, N, N);
	//convert input image real device vector to complex device vector
	real2complex<<<dimGrid,dimBlock>>>(image_dev, image_complex_dev, N, N);

	//End memory allocation of device vectors and convert/copy input image to complex device vector

	//Start creation of frequency axis
	//generate integer sequence used in creating frequency axis
	genIntSequence<<<1, N>>>(int_seq_dev, 0, N-1);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching genIntSequence!\n", cudaStatus);
        goto thickness_end;
    }
	//create omega axis
	pointwiseRealScaleRealMatrix<<<1,N>>>(freq_vector_dev, int_seq_dev, 2*PI/N, 1, N);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching pointwiseRealScaleRealMatrix!\n", cudaStatus);
        goto thickness_end;
    }
	//Shift zero to center - for even case, pull back by pi, note N is even by our assumption of powers of 2
	pointwiseAddRealConstantToRealMatrix<<<1,N>>>(freq_vector_dev, freq_vector_dev, -PI, 1, N);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addReadConstantToRealMatrix!\n", cudaStatus);
        goto thickness_end;
    }
	//convert to cyclical frequencies (hertz) and scale by pixel size
	pointwiseRealScaleRealMatrix<<<1,N>>>(freq_vector_dev, int_seq_dev, 1/(2*PI), 1, N);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching pointwiseRealScaleRealMatrix!\n", cudaStatus);
        goto thickness_end;
    }
	pointwiseRealScaleRealMatrix<<<1,N>>>(freq_vector_dev, int_seq_dev, 1/ps, 1, N);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching pointwiseRealScaleRealMatrix!\n", cudaStatus);
        goto thickness_end;
    }
	//End creation of frequency axis

	//Fourier Transform image and scale according to Paginin phase algorithm

	hipfftPlan2d(&plan, N, N, HIPFFT_C2C);
	cufftStatus = hipfftExecC2C(plan, image_complex_dev, fft_output_dev, HIPFFT_FORWARD);
	if(cufftStatus != HIPFFT_SUCCESS){
		fprintf(stderr, "hipfftExecC2C returned error code %d after attempting 2D fft!\n", cufftStatus);
        goto thickness_end;
	}
	//fft shift the spectrum of this signal
	fftShift2D<<<dimGrid,dimBlock>>>(fft_shifted_output_dev, fft_output_dev, width);
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching fftShift2D!\n", cudaStatus);
        goto thickness_end;
    }

	pointwiseRealScaleComplexMatrix<<<dimGrid,dimBlock>>>(fft_shifted_output_dev, fft_shifted_output_dev, mu/IinVal, N, N);
	//End Fourier Transform and scaling

	//Create the denominator shown in the Paganin phase algorithm
	xrayTIEHelperKernel<<<dimGrid, dimBlock>>>(denominator_dev, freq_vector_dev, N, R2, delta, Mag, mu, reg);
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching xrayTIEHelperKernel!\n", cudaStatus);
        goto thickness_end;
    }
	//End creation of denominator

	//pointwise divide, ifft, pointwise log, and inverse-mu-scaling as shown in Paganin phase algorithm
	//pointwise divide
	pointwiseDivideComplexMatrices<<<dimGrid, dimBlock>>>(fft_shifted_output_dev, fft_shifted_output_dev, denominator_dev, N, N);
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching xrayTIEHelperKernel!\n", cudaStatus);
        goto thickness_end;
    }
	//ifftshift
	fftShift2D<<<dimGrid, dimBlock>>>(ifft_shifted_input_dev, fft_shifted_output_dev, N);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching fftShift2D!\n", cudaStatus);
        goto thickness_end;
    }
	//ifft
	cufftStatus = hipfftExecC2C(plan, ifft_shifted_input_dev, ifft_output_dev, HIPFFT_BACKWARD);
	if(cufftStatus != HIPFFT_SUCCESS){
		fprintf(stderr, "hipfftExecC2C returned error code %d after attempting 2D fft!\n", cufftStatus);
        goto thickness_end;
	}
	//normalized and convert to real device vector
	float scale = 1.f / ( (float) height * (float) width );
	//convert complex to real
	complex2real_scaled<<<dimGrid, dimBlock>>>(ifft_output_dev, output_dev, N, N, scale);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching complex2real_scaled!\n", cudaStatus);
        goto thickness_end;
    }

	//take pointwise log and scale to obtain projected thickness!
	//pointwise natural log
	pointwiseNaturalLogRealMatrix<<<dimGrid, dimBlock>>>(output_dev, output_dev, N, N);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching complex2real_scaled!\n", cudaStatus);
        goto thickness_end;
    }
	//pointwise real scale
	pointwiseRealScaleRealMatrix<<<dimGrid, dimBlock>>>(output_dev, output_dev, -(1/mu), N, N);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching pointwiseRealScaleRealMatrix!\n", cudaStatus);
        goto thickness_end;
    }

	//Transfer output device vector to our host output vector and we are done!
	cudaStatus = hipMemcpy(output, output_dev, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy for output_dev failed!");
        goto thickness_end;
    }

	//destroy cufft plan and free memory allocated on device (FYI device is another name for GPU, host is CPU)
thickness_end:
	hipFree(int_seq_dev);
	hipFree(freq_vector_dev);
	hipFree(image_dev);
	hipFree(output_dev);
	hipFree(image_complex_dev);
	hipFree(fft_output_dev);
	hipFree(fft_shifted_output_dev);
	hipFree(ifft_shifted_input_dev);
	hipFree(ifft_output_dev);
	hipFree(denominator_dev);
	if(plan)
		hipfftDestroy(plan);

	//Code ends here
	std::clock_t end = std::clock();
	double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
	std::cout << "\nDone. Took " << elapsed_secs << " seconds" << std::endl;
	return cudaStatus;
}

//computes the denominators seen in Paganin phase algorithm paper
__global__
void xrayTIEHelperKernel(hipfftComplex *denominator_dev, float *freq_vector_dev, int N, float R2, float delta, float Mag, float mu, float reg)
{
	/* compute idx and idy, the location of the element in the original NxN array */
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int idy = blockIdx.y*blockDim.y+threadIdx.y;

    if ( idx < N && idy < N) {
        int index = idx + idy*N;
		denominator_dev[index].x = (R2*delta)*(((freq_vector_dev[idx]*freq_vector_dev[idx]) + (freq_vector_dev[idy]*freq_vector_dev[idy]))/Mag) + mu + reg;
		denominator_dev[index].y = 0;
    }
}