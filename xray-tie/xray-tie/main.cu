#include "hip/hip_runtime.h"
//	main.cpp
//  orginal author: Diivanand Ramalingam
//  original institution: Computational Optical Imaging Lab at UC Berkeley (Prof. Laura Waller's Lab)

#include <ctime>
#include <cstdio>
#include <cstdlib>
#include <string>
#include <iostream>
#include "tiff_io-win.h"
#include "toolbox.h"
#include "pointwise_matrix_ops.h"
#include "fourier_tools.h"

//This constant may already be in <cmath> so I should switch the using those in the code but for now I'm using these
#define PI  3.1415926535897932384626433832795028841971693993751058209749445923078164062862089986280348253421170679
//Not all GPUS can handle large block sizes which is why it's only 16 for now
#define BLOCKSIZEX 16 //should increase this to 32,64,128, etc. to see potentially better performance gains!
#define BLOCKSIZEY 16 //should increase this to 32,64,128, etc. to see potentially better performance gains!

//Global time variable
double total_time_elapsed = 0.0;

//device function delclaration
__global__ void xrayTIEHelperKernel(hipfftComplex *denominator_dev, float *freq_vector_dev, int N, float R2, float delta, float Mag, float mu, float reg);

//host function declarations
void stateArguments(float IinVal, float Mag, float R2, float mu, float delta, float ps, float reg);
hipError_t calculateThickness(float *output, float *image, int height, int width, float IinVal, float Mag, float R2, float mu, float delta, float ps, float reg);

/* Height is the number of rows (x) and Width is the number of columns (y)*/
int main(int argc, char **argv)
{
	if(argc != 13){
		printf("Incorrect number of arguments. Usage: ./tie input_folder output_folder prefix start_num end_num Iin Mag R2 mu delta ps reg\n");
		quitProgramPrompt(0);
		return 1;
	}else {
		char *srcFolder = argv[1];
		char *destFolder = argv[2];
		char *prefix = argv[3];
		int start = atoi(argv[4]);
		int end = atoi(argv[5]);
		int numFiles = end - start + 1;
		char **filenames = getFilenames(srcFolder, prefix, start, end);
		char **outfilenames = getFilenames(destFolder, prefix, start, end);
		TIFFSetWarningHandler(NULL);
		//IinVal, Mag, R2, mu, delta, ps
		float IinVal = atof(argv[6]);
		float Mag = atof(argv[7]);
		//Mag = 1.0; //Right now algorithm doesn't work for Mag other than 1.0, so for now Mag argument isn't supported.
		float R2 = atof(argv[8]);
		float mu = atof(argv[9]);
		float delta = atof(argv[10]);
		float ps = atof(argv[11]);
		float reg = atof(argv[12]);

		stateArguments(IinVal, Mag, R2, mu, delta, ps, reg);

		TiffIO* tiff_io = new TiffIO();
		int width;
		int height;
		printf("Processing Input Files: \n");
		for(int i = 0;i < numFiles; i++) {
			float **image;
			//read iamge
			image = tiff_io->read16bitImage(filenames[i], &width, &height);

			if(!image){
				printf("Error reading image\n");
			}else {
				//convert image to 1D for CUDA processing
				float *image1D = toFloatArray(image, width, height);
				
				float *image_dev = 0;
				float *output = 0;
				
				printf("\nProcessing file %s\n", filenames[i]);
				//Process Image

				//Allocate space on GPU and then transfer the input image to the GPU
				hipError_t cudaStatus;

				cudaStatus = hipMalloc((void**)&image_dev, sizeof(float) * height * width);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMalloc for image_dev failed! Error Code: %d", cudaStatus);
				}else {
					cudaStatus = hipMemcpy(image_dev, image1D, sizeof(float) * width * height, hipMemcpyHostToDevice);
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "hipMemcpy for image_dev failed! Error Code: %d", cudaStatus);
					}else{
						output = (float *) malloc(sizeof(float) * width * height);
						calculateThickness(output, image_dev, height, width, IinVal, Mag, R2, mu, delta, ps, reg);
					}
				}

				//End Processing of Image
				//convert 1D output to 2D for outputting
				float *image1DOut = 0;
				if(output){
					//printMatrix1D("output", output, height, width);
					image1DOut = output;
				}else {
					fprintf(stderr, "\n Output is NULL so using original image as output");
					image1DOut = image1D;
				}
				float **imageOut = toFloat2D(image1DOut, width, height);
				//output image
				printf("\nFile Processed. Outputting to %s\n", outfilenames[i]);
				tiff_io->write16bitImage(imageOut, outfilenames[i], width, height);
			
				//free memory
				free(image1D);
				free(output);
				free(imageOut);
				hipFree(image_dev);
				delete image;
			}
		}
		delete tiff_io;
		printf("\nTotal time to process %d images: %f seconds\n", numFiles,total_time_elapsed);
		quitProgramPrompt(true);
		return 0;
	}
}

void stateArguments(float IinVal, float Mag, float R2, float mu, float delta, float ps, float reg)
{
	std::cout << "Input Argument Values:" << std::endl;
	std::cout << "IinVal: " << IinVal << std::endl;
	std::cout << "Mag: " << Mag << std::endl;
	std::cout << "R2: " << R2 << " mm" << std::endl;
	std::cout << "mu: " << mu << " mm^-1" << std::endl;
	std::cout << "delta: " << delta << std::endl;
	std::cout << "ps: " << ps << " mm" << std::endl;
	std::cout << "reg: " << reg << std::endl;
}

/*
Calculates thickness according to Paganin Phase paper algorithm: http://www.ncbi.nlm.nih.gov/pubmed/12000561
*/
hipError_t calculateThickness(float* output, float *image_dev, int height, int width, float IinVal, float Mag, float R2, float mu, float delta, float ps, float reg)
{
	hipError_t cudaStatus;
	hipfftResult cufftStatus;
	hipfftHandle plan = 0;
	if(height != width){
		fprintf(stderr, "Only works on square matrices whose dimension is a power of two!\n", cudaStatus);
        goto thickness_end;
	}
	//declare and initialize variables used when calling CUDA kernels
	int size = height * width;
	int block_size_x = BLOCKSIZEX;
    int block_size_y = BLOCKSIZEY;
    dim3 dimBlock(block_size_x, block_size_y);
    dim3 dimGrid (height/dimBlock.x, width/dimBlock.y);
	int N = width;
	//Handle N not multiple of block_size_x or block_size_y but this shouldn't be the case since N power of 2
	//And blocksize should always be a power of 2 for both correctness and efficiency
    if (height % block_size_x !=0 ) dimGrid.x+=1;
    if (width % block_size_y !=0 ) dimGrid.y+=1;

	std::cout << "Calculating Thickness..." << std::endl;
	std::clock_t begin = std::clock();
	//Code begins here

	//declare device pointers
	float *int_seq_dev = 0;
	float *freq_vector_dev = 0;
	//float *image_dev = 0;
	float *output_dev = 0;
	hipfftComplex *image_complex_dev = 0;
	hipfftComplex *fft_output_dev = 0;
	hipfftComplex *fft_shifted_output_dev = 0;
	hipfftComplex *ifft_shifted_input_dev = 0;
	hipfftComplex *ifft_output_dev = 0;
	hipfftComplex *denominator_dev = 0;

	//Start memory allocation of device vectors and convert/copy input image to complex device vector

	//Allocate memory for 9 device vectors (potential speedup to be gained by reducing the number of device vectors used)
	cudaStatus = hipMalloc((void**)&int_seq_dev, N * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for int_seq_dev failed! Error Code: %d", cudaStatus);
        goto thickness_end;
    }
	cudaStatus = hipMalloc((void**)&freq_vector_dev, N * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for freq_vector_dev failed! Error Code: %d", cudaStatus);
        goto thickness_end;
    }
	cudaStatus = hipMalloc((void**)&output_dev, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for output_dev failed! Error Code: %d", cudaStatus);
        goto thickness_end;
    }
	cudaStatus = hipMalloc((void**)&image_complex_dev, size * sizeof(hipfftComplex));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for image_complex_dev failed! Error Code: %d", cudaStatus);
        goto thickness_end;
    }
	cudaStatus = hipMalloc((void**)&fft_output_dev, size * sizeof(hipfftComplex));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for fft_output_dev failed! Error Code: %d", cudaStatus);
        goto thickness_end;
    }
	cudaStatus = hipMalloc((void**)&fft_shifted_output_dev, size * sizeof(hipfftComplex));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for fft_shifted_output_dev failed! Error Code: %d", cudaStatus);
        goto thickness_end;
    }
	cudaStatus = hipMalloc((void**)&ifft_shifted_input_dev, size * sizeof(hipfftComplex));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for ifft_shifted_input_dev failed! Error Code: %d", cudaStatus);
        goto thickness_end;
    }
	cudaStatus = hipMalloc((void**)&ifft_output_dev, size * sizeof(hipfftComplex));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for ifft_output_dev failed! Error Code: %d", cudaStatus);
        goto thickness_end;
    }
	cudaStatus = hipMalloc((void**)&denominator_dev, size * sizeof(hipfftComplex));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc for denominator_dev failed! Error Code: %d", cudaStatus);
        goto thickness_end;
    }
	
	//scale by magnification
	//printDeviceMatrixValues("img_dev", image_dev,N,N);
	pointwiseRealScaleRealMatrix<<<dimGrid, dimBlock>>>(image_dev, image_dev, Mag*Mag, N, N);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching pointwiseRealScaleMatrix!\n", cudaStatus);
        goto thickness_end;
    }
	//printDeviceMatrixValues("img_dev_scaled", image_dev,N,N);
	//convert input image real device vector to complex device vector
	real2complex<<<dimGrid,dimBlock>>>(image_dev, image_complex_dev, N, N);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching real2complex!\n", cudaStatus);
        goto thickness_end;
    }
	//printDeviceComplexMatrixValues("img_complex_dev", image_complex_dev,N,N);
	//End memory allocation of device vectors and convert/copy input image to complex device vector

	//Start creation of frequency axis
	//generate integer sequence used in creating frequency axis
	genIntSequence<<<1, N>>>(int_seq_dev, 0, N-1);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching genIntSequence!\n", cudaStatus);
        goto thickness_end;
    }
	//printDeviceMatrixValues("int_seq_dev", int_seq_dev,1,N);
	//create omega axis
	pointwiseRealScaleRealMatrix<<<1,N>>>(freq_vector_dev, int_seq_dev, 2*PI/N, N, 1);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching pointwiseRealScaleRealMatrix!\n", cudaStatus);
        goto thickness_end;
    }
	//printDeviceMatrixValues("omega_freq_vector_dev", freq_vector_dev,1,N);
	//Shift zero to center - for even case, pull back by pi, note N is even by our assumption of powers of 2
	pointwiseAddRealConstantToRealMatrix<<<1,N>>>(freq_vector_dev, freq_vector_dev, -PI, N, 1);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addReadConstantToRealMatrix!\n", cudaStatus);
        goto thickness_end;
    }
	//convert to cyclical frequencies (hertz) and scale by pixel size
	pointwiseRealScaleRealMatrix<<<1,N>>>(freq_vector_dev, freq_vector_dev, 1/(2*PI), N, 1);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching pointwiseRealScaleRealMatrix!\n", cudaStatus);
        goto thickness_end;
    }
	pointwiseRealScaleRealMatrix<<<1,N>>>(freq_vector_dev, freq_vector_dev, 1/ps, N, 1);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching pointwiseRealScaleRealMatrix!\n", cudaStatus);
        goto thickness_end;
    }
	//printDeviceMatrixValues("freq_vector_dev after Pi shift to center", freq_vector_dev,1,N);
	//End creation of frequency axis

	//Fourier Transform image and scale according to Paginin phase algorithm

	hipfftPlan2d(&plan, N, N, HIPFFT_C2C);
	cufftStatus = hipfftExecC2C(plan, image_complex_dev, fft_output_dev, HIPFFT_FORWARD);
	if(cufftStatus != HIPFFT_SUCCESS){
		fprintf(stderr, "hipfftExecC2C returned error code %d after attempting 2D fft!\n", cufftStatus);
        goto thickness_end;
	}
	//fft shift the spectrum of this signal
	fftShift2D<<<dimGrid,dimBlock>>>(fft_shifted_output_dev, fft_output_dev, width);
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching fftShift2D!\n", cudaStatus);
        goto thickness_end;
    }
	
	//printDeviceComplexMatrixValues("fft_shifted_output_dev", fft_shifted_output_dev, N,N);
	pointwiseRealScaleComplexMatrix<<<dimGrid,dimBlock>>>(fft_shifted_output_dev, fft_shifted_output_dev, mu/IinVal, N, N);
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after pointwiseRealScaleComplexMatrix!\n", cudaStatus);
        goto thickness_end;
    }
	//printDeviceComplexMatrixValues("fft_shifted_output_dev scaled", fft_shifted_output_dev, N,N);
	//End Fourier Transform and scaling

	//Create the denominator shown in the Paganin phase algorithm
	xrayTIEHelperKernel<<<dimGrid, dimBlock>>>(denominator_dev, freq_vector_dev, N, R2, delta, Mag, mu, reg);
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching xrayTIEHelperKernel!\n", cudaStatus);
        goto thickness_end;
    }
	//printDeviceComplexMatrixValues("denominator_dev", denominator_dev, N,N);
	//End creation of denominator

	//pointwise divide, ifft, pointwise log, and inverse-mu-scaling as shown in Paganin phase algorithm
	//pointwise divide
	pointwiseDivideComplexMatrices<<<dimGrid, dimBlock>>>(fft_shifted_output_dev, fft_shifted_output_dev, denominator_dev, N, N);
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching xrayTIEHelperKernel!\n", cudaStatus);
        goto thickness_end;
    }
	//printDeviceComplexMatrixValues("inv_term (as mentioned in matlab algorithm)", fft_shifted_output_dev, N,N);
	//ifftshift
	fftShift2D<<<dimGrid, dimBlock>>>(ifft_shifted_input_dev, fft_shifted_output_dev, N);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching fftShift2D!\n", cudaStatus);
        goto thickness_end;
    }
	//ifft
	cufftStatus = hipfftExecC2C(plan, ifft_shifted_input_dev, ifft_output_dev, HIPFFT_BACKWARD);
	if(cufftStatus != HIPFFT_SUCCESS){
		fprintf(stderr, "hipfftExecC2C returned error code %d after attempting 2D fft!\n", cufftStatus);
        goto thickness_end;
	}
	//normalized and convert to real device vector
	float scale = 1.f / ( (float) height * (float) width );
	//convert complex to real
	complex2real_scaled<<<dimGrid, dimBlock>>>(ifft_output_dev, output_dev, N, N, scale);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching complex2real_scaled!\n", cudaStatus);
        goto thickness_end;
    }
	//printDeviceMatrixValues("ifftReal", output_dev, N,N);
	//take pointwise log and scale to obtain projected thickness!
	//pointwise natural log
	pointwiseNaturalLogRealMatrix<<<dimGrid, dimBlock>>>(output_dev, output_dev, N, N);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching complex2real_scaled!\n", cudaStatus);
        goto thickness_end;
    }
	//printDeviceMatrixValues("logOutput (like in matlab algorithm)", output_dev, N,N);
	//pointwise real scale
	pointwiseRealScaleRealMatrix<<<dimGrid, dimBlock>>>(output_dev, output_dev, -(1/mu), N, N);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching pointwiseRealScaleRealMatrix!\n", cudaStatus);
        goto thickness_end;
    }
	//printDeviceMatrixValues("output", output_dev, N,N);
	//Transfer output device vector to our host output vector and we are done!
	cudaStatus = hipMemcpy(output, output_dev, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy for output_dev failed!");
        goto thickness_end;
    }

	//destroy cufft plan and free memory allocated on device (FYI device is another name for GPU, host is CPU)
thickness_end:
	hipFree(int_seq_dev);
	hipFree(freq_vector_dev);
	hipFree(output_dev);
	hipFree(image_complex_dev);
	hipFree(fft_output_dev);
	hipFree(fft_shifted_output_dev);
	hipFree(ifft_shifted_input_dev);
	hipFree(ifft_output_dev);
	hipFree(denominator_dev);
	if(plan)
		hipfftDestroy(plan);

	//Code ends here
	std::clock_t end = std::clock();
	double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
	std::cout << "\nDone. Took " << elapsed_secs << " seconds" << std::endl;
	total_time_elapsed += elapsed_secs;
	return cudaStatus;
}

//computes the denominators seen in Paganin phase algorithm paper
__global__
void xrayTIEHelperKernel(hipfftComplex *denominator_dev, float *freq_vector_dev, int N, float R2, float delta, float Mag, float mu, float reg)
{
	/* compute idx and idy, the location of the element in the original NxN array */
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int idy = blockIdx.y*blockDim.y+threadIdx.y;

    if ( idx < N && idy < N) {
        int index = idx + idy*N;
		denominator_dev[index].x = (R2*delta)*(((freq_vector_dev[idx]*freq_vector_dev[idx]) + (freq_vector_dev[idy]*freq_vector_dev[idy]))/Mag) + mu + reg;
		denominator_dev[index].y = 0;
    }
}